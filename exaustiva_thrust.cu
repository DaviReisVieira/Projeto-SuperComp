#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>

#include <thrust/host_vector.h>
#include <thrust/sequence.h>
#include <thrust/device_vector.h>
#include <thrust/functional.h>
#include <thrust/copy.h>

using namespace std;

struct Filme
{
  int id;
  int inicio;
  int fim;
  int categoria;
};

void preenche_horarios(int &horarios_disponiveis, int inicio, int fim)
{
  for (int i = inicio; i < fim; i++)
  {
    horarios_disponiveis |= (1 << i);
  }
}

struct busca_exaustiva_gpu
{
  int n_filmes;
  int m_categorias;

  int *disponibilidade_categoria;
  int *horarios_filmes;
  int *categoria_filmes;

  busca_exaustiva_gpu(int n_filmes_,
                      int m_categorias_,
                      int *disponibilidade_categoria_,
                      int *horarios_filmes_,
                      int *categoria_filmes_)
      : n_filmes(n_filmes_),
        m_categorias(m_categorias_),
        disponibilidade_categoria(disponibilidade_categoria_),
        horarios_filmes(horarios_filmes_),
        categoria_filmes(categoria_filmes_)
  {
  }

  __device__ int operator()(const int &config)
  {
    int horarios_disponiveis = 0;
    int categorias_vistas[16];

    for (int i = 0; i < m_categorias; i++)
    {
      categorias_vistas[i] = disponibilidade_categoria[i];
    }

    int num_filmes = 0;
    for (int i = 0; i < n_filmes; i++)
    {
      if (config & (1 << i))
      {
        if (categorias_vistas[categoria_filmes[i] - 1] > 0)
        {
          int horario_analisado = horarios_disponiveis & horarios_filmes[i];
          if (horario_analisado != 0)
            return false;

          categorias_vistas[categoria_filmes[i] - 1]--;
          horarios_disponiveis |= horarios_filmes[i];
          num_filmes++;
        }
      }
    }

    return num_filmes;
  }
};

int main()
{
  int n, m;
  cin >> n >> m;

  vector<Filme> vetor_filmes;
  thrust::host_vector<int> disponibilidade_categoria(m);

  for (int i = 0; i < m; i++)
  {
    cin >> disponibilidade_categoria[i];
  }

  for (int i = 0; i < n; i++)
  {
    int inicio, fim, categoria;
    cin >> inicio >> fim >> categoria;

    if (inicio > fim)
    {
      if (fim == 0)
      {
        fim = 24;
      }
      else if (inicio == -1 || fim == -1)
      {
        continue;
      }
      else
      {
        continue;
      }
    }

    Filme filme;
    filme.id = i + 1;
    filme.inicio = inicio;
    filme.fim = fim;
    filme.categoria = categoria;

    vetor_filmes.push_back(filme);
  }

  int n_filmes = vetor_filmes.size();

  thrust::host_vector<int> categoria_filmes(n_filmes);
  thrust::host_vector<int> horarios_filmes_cpu(n_filmes);

  for (int i = 0; i < n_filmes; i++)
  {
    horarios_filmes_cpu[i] = 0;
    preenche_horarios(horarios_filmes_cpu[i],
                      vetor_filmes[i].inicio,
                      vetor_filmes[i].fim);
    categoria_filmes[i] = vetor_filmes[i].categoria;
  }

  thrust::device_vector<int> vetor_possibilidades_gpu(pow(2, n_filmes));

  thrust::sequence(vetor_possibilidades_gpu.begin(), vetor_possibilidades_gpu.end());

  thrust::device_vector<int> disponibilidade_categoria_gpu(disponibilidade_categoria);

  thrust::device_vector<int> horarios_filmes_gpu(horarios_filmes_cpu);
  thrust::device_vector<int> categoria_filmes_gpu(categoria_filmes);

  thrust::transform(
      vetor_possibilidades_gpu.begin(),
      vetor_possibilidades_gpu.end(),
      vetor_possibilidades_gpu.begin(),
      busca_exaustiva_gpu(n_filmes,
                          m,
                          raw_pointer_cast(disponibilidade_categoria_gpu.data()),
                          raw_pointer_cast(horarios_filmes_gpu.data()),
                          raw_pointer_cast(categoria_filmes_gpu.data())));

  thrust::host_vector<int> config_vector_cpu_final = vetor_possibilidades_gpu;

  int max_count = *thrust::max_element(config_vector_cpu_final.begin(), config_vector_cpu_final.end());

  cout << max_count << endl;

  int max_config = -1;
  for (int i = 0; i < config_vector_cpu_final.size(); i++)
  {
    if (config_vector_cpu_final[i] == max_count)
    {
      max_config = i;
      break;
    }
  }

  for (int i = 0; i < n_filmes; i++)
  {
    if (max_config & (1 << i))
    {
      cout << vetor_filmes[i].id << " " << vetor_filmes[i].inicio << " " << vetor_filmes[i].fim << " " << vetor_filmes[i].categoria << endl;
    }
  }

  return 0;
}